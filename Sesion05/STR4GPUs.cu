#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#ifndef SIZE
#define SIZE 32
#endif

// Kernel Matriz por Matriz
// C(NxM) <- A(NxP) * B (PxM)

__global__ void KernelMM(int N, int M, int P, float *A, float *B, float *C) {
//__global__ void KernelSxS (int N, int M, int P, float *A, float *B, float *C) {

  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < P; m=m+SIZE) {
    sA[ty][tx] = A[row*P + m + tx];
    sB[ty][tx] = B[col + (m + ty)*M];
    __syncthreads();
    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  C[row*M+col] = tmp;
}


// Matriz por Matriz
// C(NxM) <- A(NxP) * B (PxM)
// Usaremos siempre N, M, P multiplos de SIZE

__global__ void Kernel1x1 (int N, int M, int P, float *A, float *B, float *C) {
//__global__ void KernelMM(int N, int M, int P, float *A, float *B, float *C) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  float tmp = 0.0;
  for (int k=0; k<P; k++)
    tmp += A[row*P+k] * B[k*M+col];

  //if (row < N && col < M) 
    C[row*M+col] = tmp;
}

// STRASSEN

// Càlcul matrius M

// M1 = (A1,1 + A2,2) * (B1,1 + B2,2)
__global__ void KernelM1(int N, float *A11, float *A22, float *B11, float *B22, float *M1) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A11+A22 i B11+B22 a la memòria compartida
    sA[ty][tx] = A11[row*N + m + tx] + A22[row*N + m + tx];
    sB[ty][tx] = B11[(m + ty)*N + col] + B22[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M1[row*N+col] = tmp;
}

// M2 = (A2,1 + A2,2) * B1,1
__global__ void KernelM2(int N, float *A21, float *A22, float *B11, float *M2) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A21+A22 i B11 a la memòria compartida
    sA[ty][tx] = A21[row*N + m + tx] + A22[row*N + m + tx];
    sB[ty][tx] = B11[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M2[row*N+col] = tmp;
}

// M3 = A1,1 * (B1,2 − B2,2)
__global__ void KernelM3(int N, float *A11, float *B12, float *B22, float *M3) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A11 i B12-B22 a la memòria compartida
    sA[ty][tx] = A11[row*N + m + tx];
    sB[ty][tx] = B12[(m + ty)*N + col] - B22[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M3[row*N+col] = tmp;
}

// M4 = A2,2 * (B2,1 − B1,1)
__global__ void KernelM4(int N, float *A22, float *B21, float *B11, float *M4) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A22 i B21-B11 a la memòria compartida
    sA[ty][tx] = A22[row*N + m + tx];
    sB[ty][tx] = B21[(m + ty)*N + col] - B11[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M4[row*N+col] = tmp;
}

// M5 = (A1,1 + A1,2) * B2,2
__global__ void KernelM5(int N, float *A11, float *A12, float *B22, float *M5) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A11+A12 i B22 a la memòria compartida
    sA[ty][tx] = A11[row*N + m + tx] + A12[row*N + m + tx];
    sB[ty][tx] = B22[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M5[row*N+col] = tmp;
}

// M6 = (A2,1 − A1,1) * (B1,1 + B1,2)
__global__ void KernelM6(int N, float *A21, float *A11, float *B11, float *B12, float *M6) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A21-A11 i B11+B12 a la memòria compartida
    sA[ty][tx] = A21[row*N + m + tx] - A11[row*N + m + tx];
    sB[ty][tx] = B11[(m + ty)*N + col] + B12[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M6[row*N+col] = tmp;
}

// M7 = (A1,2 − A2,2) * (B2,1 + B2,2)
__global__ void KernelM7(int N, float *A12, float *A22, float *B21, float *B22, float *M7) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < N; m=m+SIZE) {
    // Carreguem A12-A22 i B21+B22 a la memòria compartida
    sA[ty][tx] = A12[row*N + m + tx] - A22[row*N + m + tx];
    sB[ty][tx] = B21[(m + ty)*N + col] + B22[(m + ty)*N + col];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];
    __syncthreads();
  }
  M7[row*N+col] = tmp;
}

// Càlcul matrius C

// C1,1 = M1 + M4 − M5 + M7
__global__ void KernelC11(int N, float *M1, float *M4, float *M5, float *M7, float *C11) {
  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  if (row < N && col < N) {
    int idx = row * N + col;
    C11[idx] = M1[idx] + M4[idx] - M5[idx] + M7[idx];
  }
}

// C1,2 = M3 + M5
__global__ void KernelC12(int N, float *M3, float *M5, float *C12) {
  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  if (row < N && col < N) {
    int idx = row * N + col;
    C12[idx] = M3[idx] + M5[idx];
  }
}

// C2,1 = M2 + M4
__global__ void KernelC21(int N, float *M2, float *M4, float *C21) {
  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  if (row < N && col < N) {
    int idx = row * N + col;
    C21[idx] = M2[idx] + M4[idx];
  }
}

// C2,2 = M1 − M2 + M3 + M6
__global__ void KernelC22(int N, float *M1, float *M2, float *M3, float *M6, float *C22) {
  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  if (row < N && col < N) {
    int idx = row * N + col;
    C22[idx] = M1[idx] - M2[idx] + M3[idx] + M6[idx];
  }
}

void InitM(int N, int M, float *Mat);
int TestMM(int N, int M, int P, float *A, float *B, float *C);

int nTest = 0;

// Invocacion:
// ./ejecutable TAM test
// TAM es el la dimension de las matrices
// test == 'Y', comprueba que el resultado sea correcto
// test == 'N', NO comprueba que el resultado (Util para tomar tiempos)
// Por defecto, N = 1024, test == 'N'

int main(int argc, char** argv) {
  unsigned int N;
  unsigned int numBytes, numBytesHalf;
  unsigned int nBlocks, nThreads;

  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;
  hipEvent_t X1, X2, X3;

  // Matrius host
  float *hA, *hB, *hC, *hCCheck;

  // Matrius i submatrius de GPU 0 (calcularà C11)
  float *dA0, *dB0;
  float *dA0_11, *dA0_22;
  float *dB0_11, *dB0_21, *dB0_22;
  float *dC0_11;
  float *dM0_1, *dM0_4, *dM0_5, *dM0_7;

  // Matrius i submatrius de GPU 1 (calcularà C12)
  float *dA1, *dB1;
  float *dA1_11, *dA1_12;
  float *dB1_12, *dB1_22;
  float *dC1_12;
  float *dM1_3, *dM1_5;

  // Matrius i submatrius de GPU 2 (calcularà C21)
  float *dA2, *dB2;
  float *dA2_21, *dA2_22;
  float *dB2_11, *dB2_21;
  float *dC2_21;
  float *dM2_2, *dM2_4;

  // Matrius i submatrius de GPU 3 (calcularà C22)
  float *dA3, *dB3;
  float *dA3_11, *dA3_21, *dA3_22;
  float *dB3_11, *dB3_12, *dB3_22;
  float *dC3_22;
  float *dM3_1, *dM3_2, *dM3_3, *dM3_6;

  int count;
  char test;

  // Dimension de les matrius NxN i comprovació del resultat
  if (argc == 1)      { test = 'N'; N = 1024; }
  else if (argc == 2) { test = 'N'; N = atoi(argv[1]); }
  else if (argc == 3) { test = *argv[2]; N = atoi(argv[1]); }
  else { printf("Usage: ./exe TAM test\n"); exit(0); }

  // Comprovem que la dimensió sigui potència de 2
  if ((N & (N-1)) != 0) {
    printf("La dimensió ha de ser potència de 2\n");
    exit(0);
  }

  // número de Threads en cada dimensió 
  nThreads = SIZE;

  // número de Blocks en cada dimensió (per a mida N/2)
  nBlocks = (N/2)/nThreads;

  numBytes = N * N * sizeof(float);
  numBytesHalf = (N/2) * (N/2) * sizeof(float);

  dim3 dimGrid(nBlocks, nBlocks, 1);
  dim3 dimBlock(nThreads, nThreads, 1);

  hipGetDeviceCount(&count);
  if (count < 4) { printf("No hay suficientes GPUs\n"); exit(0); }

  // Obtenim memòria al host
  hipHostMalloc((float**)&hA, numBytes);
  hipHostMalloc((float**)&hB, numBytes);
  hipHostMalloc((float**)&hC, numBytes);
  hipHostMalloc((float**)&hCCheck, numBytes);

  // Inicialitzem les matrius
  InitM(N, N, hA);
  InitM(N, N, hB);

  // Definim mida de mitja matriu
  int half_N = N/2;

  // Configuració GPU 0 (C11)
  hipSetDevice(0);
  hipMalloc((float**)&dA0, numBytes);
  hipMalloc((float**)&dB0, numBytes);

  // Només les submatrius que necessitem per C11
  hipMalloc((float**)&dA0_11, numBytesHalf);
  hipMalloc((float**)&dA0_22, numBytesHalf);
  hipMalloc((float**)&dB0_11, numBytesHalf);
  hipMalloc((float**)&dB0_21, numBytesHalf);
  hipMalloc((float**)&dB0_22, numBytesHalf);

  hipMalloc((float**)&dC0_11, numBytesHalf);

  hipMalloc((float**)&dM0_1, numBytesHalf);
  hipMalloc((float**)&dM0_4, numBytesHalf);
  hipMalloc((float**)&dM0_5, numBytesHalf);
  hipMalloc((float**)&dM0_7, numBytesHalf);

  // Configuració GPU 1 (C12)
  hipSetDevice(1);
  hipMalloc((float**)&dA1, numBytes);
  hipMalloc((float**)&dB1, numBytes);

  // Només les submatrius que necessitem per C12
  hipMalloc((float**)&dA1_11, numBytesHalf);
  hipMalloc((float**)&dA1_12, numBytesHalf);
  hipMalloc((float**)&dB1_12, numBytesHalf);
  hipMalloc((float**)&dB1_22, numBytesHalf);

  hipMalloc((float**)&dC1_12, numBytesHalf);

  hipMalloc((float**)&dM1_3, numBytesHalf);
  hipMalloc((float**)&dM1_5, numBytesHalf);

  hipEventCreate(&X1);

  // Configuració GPU 2 (C21)
  hipSetDevice(2);
  hipMalloc((float**)&dA2, numBytes);
  hipMalloc((float**)&dB2, numBytes);

  // Només les submatrius que necessitem per C21
  hipMalloc((float**)&dA2_21, numBytesHalf);
  hipMalloc((float**)&dA2_22, numBytesHalf);
  hipMalloc((float**)&dB2_11, numBytesHalf);
  hipMalloc((float**)&dB2_21, numBytesHalf);

  hipMalloc((float**)&dC2_21, numBytesHalf);

  hipMalloc((float**)&dM2_2, numBytesHalf);
  hipMalloc((float**)&dM2_4, numBytesHalf);

  hipEventCreate(&X2);

  // Configuració GPU 3 (C22)
  hipSetDevice(3);
  hipMalloc((float**)&dA3, numBytes);
  hipMalloc((float**)&dB3, numBytes);

  // Només les submatrius que necessitem per C22
  hipMalloc((float**)&dA3_11, numBytesHalf);
  hipMalloc((float**)&dA3_21, numBytesHalf);
  hipMalloc((float**)&dA3_22, numBytesHalf);
  hipMalloc((float**)&dB3_11, numBytesHalf);
  hipMalloc((float**)&dB3_12, numBytesHalf);
  hipMalloc((float**)&dB3_22, numBytesHalf);

  hipMalloc((float**)&dC3_22, numBytesHalf);

  hipMalloc((float**)&dM3_1, numBytesHalf);
  hipMalloc((float**)&dM3_2, numBytesHalf);
  hipMalloc((float**)&dM3_3, numBytesHalf);
  hipMalloc((float**)&dM3_6, numBytesHalf);

  hipEventCreate(&X3);

  // Tornem a la GPU 0 per començar els càlculs
  hipSetDevice(0);
  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  // Copiem les matrius A i B al device per a tots els GPUs
  hipSetDevice(0);
  hipMemcpy(dA0, hA, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(dB0, hB, numBytes, hipMemcpyHostToDevice);

  hipSetDevice(1);
  hipMemcpy(dA1, hA, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(dB1, hB, numBytes, hipMemcpyHostToDevice);

  hipSetDevice(2);
  hipMemcpy(dA2, hA, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(dB2, hB, numBytes, hipMemcpyHostToDevice);

  hipSetDevice(3);
  hipMemcpy(dA3, hA, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(dB3, hB, numBytes, hipMemcpyHostToDevice);

  hipSetDevice(0);
  hipEventRecord(E0, 0);

  // Iniciem el cronòmetre per mesurar el temps dels kernels
  hipEventRecord(E1, 0);

  // GPU 0: Extreure les submatrius A i B necessàries per C11
  hipSetDevice(0);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&dA0_11[i*half_N], &hA[i*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dA0_22[i*half_N], &hA[(i+half_N)*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(&dB0_11[i*half_N], &hB[i*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dB0_21[i*half_N], &hB[(i+half_N)*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dB0_22[i*half_N], &hB[(i+half_N)*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
  }

  // GPU 1: Extreure les submatrius A i B necessàries per C12
  hipSetDevice(1);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&dA1_11[i*half_N], &hA[i*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dA1_12[i*half_N], &hA[i*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(&dB1_12[i*half_N], &hB[i*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dB1_22[i*half_N], &hB[(i+half_N)*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
  }

  // GPU 2: Extreure les submatrius A i B necessàries per C21
  hipSetDevice(2);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&dA2_21[i*half_N], &hA[(i+half_N)*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dA2_22[i*half_N], &hA[(i+half_N)*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(&dB2_11[i*half_N], &hB[i*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dB2_21[i*half_N], &hB[(i+half_N)*N], half_N*sizeof(float), hipMemcpyHostToDevice);
  }

  // GPU 3: Extreure les submatrius A i B necessàries per C22
  hipSetDevice(3);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&dA3_11[i*half_N], &hA[i*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dA3_21[i*half_N], &hA[(i+half_N)*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dA3_22[i*half_N], &hA[(i+half_N)*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(&dB3_11[i*half_N], &hB[i*N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dB3_12[i*half_N], &hB[i*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(&dB3_22[i*half_N], &hB[(i+half_N)*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
  }
  // GPU 0: Calculem les matrius M necessàries per C11
  hipSetDevice(0);
  // M1 = (A1,1 + A2,2) · (B1,1 + B2,2)
  KernelM1<<<dimGrid, dimBlock>>>(half_N, dA0_11, dA0_22, dB0_11, dB0_22, dM0_1);
  // M4 = A2,2 · (B2,1 − B1,1)
  KernelM4<<<dimGrid, dimBlock>>>(half_N, dA0_22, dB0_21, dB0_11, dM0_4);
  // Per a M5, necessitem A1,2, però no el tenim a la GPU 0 -->> Decisió Gian 1: Copiar A1,2 de la host memory a la GPU0
  float *dA0_12;
  hipMalloc((float**)&dA0_12, numBytesHalf);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&dA0_12[i*half_N], &hA[i*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
  }
  // M5 = (A1,1 + A1,2) · B2,2
  KernelM5<<<dimGrid, dimBlock>>>(half_N, dA0_11, dA0_12, dB0_22, dM0_5);
  hipFree(dA0_12);

  // M7 = (A1,2 − A2,2) · (B2,1 + B2,2)
  hipMalloc((float**)&dA0_12, numBytesHalf);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&dA0_12[i*half_N], &hA[i*N+half_N], half_N*sizeof(float), hipMemcpyHostToDevice);
  }
  KernelM7<<<dimGrid, dimBlock>>>(half_N, dA0_12, dA0_22, dB0_21, dB0_22, dM0_7);
  hipFree(dA0_12);

  // GPU 1: Calculem les matrius M necessàries per C12
  hipSetDevice(1);
  // M3 = A1,1 · (B1,2 − B2,2)
  KernelM3<<<dimGrid, dimBlock>>>(half_N, dA1_11, dB1_12, dB1_22, dM1_3);
  // M5 = (A1,1 + A1,2) · B2,2
  KernelM5<<<dimGrid, dimBlock>>>(half_N, dA1_11, dA1_12, dB1_22, dM1_5);

  // GPU 2: Calculem les matrius M necessàries per C21
  hipSetDevice(2);
  // M2 = (A2,1 + A2,2) · B1,1
  KernelM2<<<dimGrid, dimBlock>>>(half_N, dA2_21, dA2_22, dB2_11, dM2_2);
  // M4 = A2,2 · (B2,1 − B1,1)
  KernelM4<<<dimGrid, dimBlock>>>(half_N, dA2_22, dB2_21, dB2_11, dM2_4);

  // GPU 3: Calculem les matrius M necessàries per C22
  hipSetDevice(3);
  // M1 = (A1,1 + A2,2) · (B1,1 + B2,2)
  KernelM1<<<dimGrid, dimBlock>>>(half_N, dA3_11, dA3_22, dB3_11, dB3_22, dM3_1);
  // M2 = (A2,1 + A2,2) · B1,1
  KernelM2<<<dimGrid, dimBlock>>>(half_N, dA3_21, dA3_22, dB3_11, dM3_2);
  // M3 = A1,1 · (B1,2 − B2,2)
  KernelM3<<<dimGrid, dimBlock>>>(half_N, dA3_11, dB3_12, dB3_22, dM3_3);
  // M6 = (A2,1 − A1,1) · (B1,1 + B1,2)
  KernelM6<<<dimGrid, dimBlock>>>(half_N, dA3_21, dA3_11, dB3_11, dB3_12, dM3_6);

  // Calculem les matrius C

  // GPU 0: Calculem C11
  hipSetDevice(0);
  // C1,1 = M1 + M4 − M5 + M7
  KernelC11<<<dimGrid, dimBlock>>>(half_N, dM0_1, dM0_4, dM0_5, dM0_7, dC0_11);

  // GPU 1: Calculem C12
  hipSetDevice(1);
  // C1,2 = M3 + M5
  KernelC12<<<dimGrid, dimBlock>>>(half_N, dM1_3, dM1_5, dC1_12);

  // GPU 2: Calculem C21
  hipSetDevice(2);
  // C2,1 = M2 + M4
  KernelC21<<<dimGrid, dimBlock>>>(half_N, dM2_2, dM2_4, dC2_21);

  // GPU 3: Calculem C22
  hipSetDevice(3);
  // C2,2 = M1 − M2 + M3 + M6
  KernelC22<<<dimGrid, dimBlock>>>(half_N, dM3_1, dM3_2, dM3_3, dM3_6, dC3_22);

  // Finalitzem el cronòmetre dels kernels
  hipSetDevice(0);
  hipEventRecord(E2, 0);

  // Unir les submatrius C per obtenir el resultat final
  // Copiem les dades cap al host
  hipSetDevice(0);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&hC[i*N], &dC0_11[i*half_N], half_N*sizeof(float), hipMemcpyDeviceToHost);
  }

  hipSetDevice(1);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&hC[i*N+half_N], &dC1_12[i*half_N], half_N*sizeof(float), hipMemcpyDeviceToHost);
  }
  hipEventRecord(X1, 0);

  hipSetDevice(2);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&hC[(i+half_N)*N], &dC2_21[i*half_N], half_N*sizeof(float), hipMemcpyDeviceToHost);
  }
  hipEventRecord(X2, 0);

  hipSetDevice(3);
  for (int i = 0; i < half_N; i++) {
    hipMemcpy(&hC[(i+half_N)*N+half_N], &dC3_22[i*half_N], half_N*sizeof(float), hipMemcpyDeviceToHost);
  }
  hipEventRecord(X3, 0);

  hipSetDevice(0);
  hipEventSynchronize(X1);
  hipEventSynchronize(X2);
  hipEventSynchronize(X3);
  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  // Alliberem memòria al device
  hipSetDevice(0);
  hipFree(dA0); hipFree(dB0);
  hipFree(dA0_11); hipFree(dA0_22);
  hipFree(dB0_11); hipFree(dB0_21); hipFree(dB0_22);
  hipFree(dC0_11);
  hipFree(dM0_1); hipFree(dM0_4); hipFree(dM0_5); hipFree(dM0_7);

  hipSetDevice(1);
  hipFree(dA1); hipFree(dB1);
  hipFree(dA1_11); hipFree(dA1_12);
  hipFree(dB1_12); hipFree(dB1_22);
  hipFree(dC1_12);
  hipFree(dM1_3); hipFree(dM1_5);

  hipSetDevice(2);
  hipFree(dA2); hipFree(dB2);
  hipFree(dA2_21); hipFree(dA2_22);
  hipFree(dB2_11); hipFree(dB2_21);
  hipFree(dC2_21);
  hipFree(dM2_2); hipFree(dM2_4);

  hipSetDevice(3);
  hipFree(dA3); hipFree(dB3);
  hipFree(dA3_11); hipFree(dA3_21); hipFree(dA3_22);
  hipFree(dB3_11); hipFree(dB3_12); hipFree(dB3_22);
  hipFree(dC3_22);
  hipFree(dM3_1); hipFree(dM3_2); hipFree(dM3_3); hipFree(dM3_6);

  hipSetDevice(0);
  hipEventElapsedTime(&TiempoTotal, E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);

  printf("\nKERNEL Strassen 4 GPUs - Producto Matrices\n");
  printf("Dimensiones: %dx%d\n", N, N);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
  printf("Tiempo Global: %4.6f milseg\n", TiempoTotal);
  printf("Tiempo Kernel: %4.6f milseg\n", TiempoKernel);
  printf("Rendimiento Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
  printf("Rendimiento Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

  hipSetDevice(0); hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);
  hipSetDevice(1); hipEventDestroy(X1);
  hipSetDevice(2); hipEventDestroy(X2);
  hipSetDevice(3); hipEventDestroy(X3);

  if (test == 'N')
    printf("NO TEST\n");
  else {
    // Calculem el producte a la CPU per verificar
    float *cpuC = (float*)malloc(numBytes);

    for (int i = 0; i < N; i++) {
      for (int j = 0; j < N; j++) {
        cpuC[i*N+j] = 0.0;
        for (int k = 0; k < N; k++)
          cpuC[i*N+j] += hA[i*N+k] * hB[k*N+j];
      }
    }

    // Comparem els resultats
    bool correct = true;
    for (int i = 0; i < N*N; i++) {
      if (fabs(cpuC[i] - hC[i]) > 0.0001) {
        correct = false;
        printf("Error a la posició %d: CPU=%f, GPU=%f\n", i, cpuC[i], hC[i]);
        break;
      }
    }

    if (correct)
      printf("TEST PASS\n");
    else
      printf("TEST FAIL\n");

    free(cpuC);
  }

  hipHostFree(hA); hipHostFree(hB); hipHostFree(hC); hipHostFree(hCCheck);

}

void InitM(int N, int M, float *Mat) {
   int i;
   for (i=0; i<N*M; i++) 
     Mat[i] = rand() / (float) RAND_MAX;
   
}

int error(float a, float b) {
  float tmp;

  tmp = abs(a-b) / abs(min(a,b));

  if (isnan(tmp) || tmp > 0.0001) return 1;
  else  return 0;

}

int TestMM(int N, int M, int P, float *A, float *B, float *C) {
   int i, j, k;
   float tmp;
   printf("Pass %d\n", nTest); nTest++;
   for (i=0; i<N; i++)
     for (j=0; j<M; j++) {
       tmp = 0.0;
       for (k=0; k<P; k++) 
         tmp = tmp + A[i*P+k] * B[k*M+j]; 
       if (error(tmp, C[i*M+j])) {
         printf ("%d:%d: %f - %f = %f \n", i, j, tmp, C[i*M+j], abs(tmp - C[i*M+j]));
         return 0;
       }
     }
   
   return 1;
}

