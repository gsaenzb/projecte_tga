#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#ifndef SIZE
#define SIZE 32
#endif

#ifndef PINNED
#define PINNED 0
#endif

// Kernel per a la multiplicació de matrius en blocs (el mateix que Kernel10)
__global__ void KernelMult(int N, int M, int P, float *A, float *B, float *C) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  float tmp = 0.0;
  for (int m=0; m < P; m=m+SIZE) {
    sA[ty][tx] = A[row*P + m + tx];
    sB[ty][tx] = B[col + (m + ty)*M];
    __syncthreads();

    for (int k=0; k<SIZE; k++)
      tmp += sA[ty][k] * sB[k][tx];

    __syncthreads();
  }
  C[row*M+col] = tmp;
}

// Kernel per a l'addició de matrius
__global__ void KernelAdd(int N, int M, float *A, float *B, float *C) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  // Càrrega de dades a la memòria compartida
  if(row < N && col < M) {
    sA[ty][tx] = A[row*M + col];
    sB[ty][tx] = B[row*M + col];
  } else {
    sA[ty][tx] = 0.0;
    sB[ty][tx] = 0.0;
  }
  __syncthreads();

  // Realitzem l'addició
  if(row < N && col < M) {
    C[row*M + col] = sA[ty][tx] + sB[ty][tx];
  }
}

// Kernel per a la subtracció de matrius
__global__ void KernelSub(int N, int M, float *A, float *B, float *C) {
  __shared__ float sA[SIZE][SIZE];
  __shared__ float sB[SIZE][SIZE];

  int bx = blockIdx.x;  int by = blockIdx.y;
  int tx = threadIdx.x; int ty = threadIdx.y;
  int row = by * SIZE + ty;
  int col = bx * SIZE + tx;

  // Càrrega de dades a la memòria compartida
  if(row < N && col < M) {
    sA[ty][tx] = A[row*M + col];
    sB[ty][tx] = B[row*M + col];
  } else {
    sA[ty][tx] = 0.0;
    sB[ty][tx] = 0.0;
  }
  __syncthreads();

  // Realitzem la subtracció
  if(row < N && col < M) {
    C[row*M + col] = sA[ty][tx] - sB[ty][tx];
  }
}

// Funció per inicialitzar matrius
void InitM(int N, int M, float *Mat) {
  int i;
  for (i=0; i<N*M; i++) 
    Mat[i] = rand() / (float) RAND_MAX;
}

// Funció per comprovar si hi ha error en el resultat
int error(float a, float b) {
  float tmp;
  tmp = abs(a-b) / abs(min(a,b));
  if (isnan(tmp) || tmp > 0.0001) return 1;
  else return 0;
}

// Funció per comprovar el producte de matrius
int TestMM(int N, int M, int P, float *A, float *B, float *C) {
  int i, j, k;
  float tmp;
  for (i=0; i<N; i++)
    for (j=0; j<M; j++) {
      tmp = 0.0;
      for (k=0; k<P; k++) 
        tmp = tmp + A[i*P+k] * B[k*M+j]; 
      if (error(tmp, C[i*M+j])) {
        printf ("%d:%d: %f - %f = %f \n", i, j, tmp, C[i*M+j], abs(tmp - C[i*M+j]));
        return 0;
      }
    }
  
  return 1;
}

// Funció per dividir una matriu en submatrius
void partitionMatrix(float *mat, float *submat11, float *submat12, float *submat21, float *submat22, int n) {
  int halfn = n/2;
  int size = halfn * halfn;
  
  for(int i = 0; i < halfn; i++) {
    for(int j = 0; j < halfn; j++) {
      // Submatriu 11 (superior esquerra)
      submat11[i*halfn + j] = mat[i*n + j];
      
      // Submatriu 12 (superior dreta)
      submat12[i*halfn + j] = mat[i*n + j + halfn];
      
      // Submatriu 21 (inferior esquerra)
      submat21[i*halfn + j] = mat[(i + halfn)*n + j];
      
      // Submatriu 22 (inferior dreta)
      submat22[i*halfn + j] = mat[(i + halfn)*n + j + halfn];
    }
  }
}

// Funció per reunir les submatrius en una matriu completa
void joinMatrix(float *mat, float *submat11, float *submat12, float *submat21, float *submat22, int n) {
  int halfn = n/2;
  int size = halfn * halfn;
  
  for(int i = 0; i < halfn; i++) {
    for(int j = 0; j < halfn; j++) {
      // Submatriu 11 (superior esquerra)
      mat[i*n + j] = submat11[i*halfn + j];
      
      // Submatriu 12 (superior dreta)
      mat[i*n + j + halfn] = submat12[i*halfn + j];
      
      // Submatriu 21 (inferior esquerra)
      mat[(i + halfn)*n + j] = submat21[i*halfn + j];
      
      // Submatriu 22 (inferior dreta)
      mat[(i + halfn)*n + j + halfn] = submat22[i*halfn + j];
    }
  }
}

// Funció principal per a l'execució
int main(int argc, char** argv)
{
  unsigned int N, size;
  unsigned int numBytes, numBytesSubmatrix;
  unsigned int nBlocks, nThreads;
 
  float TiempoTotal, TiempoKernel;
  hipEvent_t E0, E1, E2, E3;

  float *h_A, *h_B, *h_C;
  float *d_A, *d_B, *d_C;
  
  // Matrius temporals per a Strassen (device)
  float *d_M1, *d_M2, *d_M3, *d_M4, *d_M5, *d_M6, *d_M7;
  float *d_A11, *d_A12, *d_A21, *d_A22;
  float *d_B11, *d_B12, *d_B21, *d_B22;
  float *d_C11, *d_C12, *d_C21, *d_C22;
  float *d_temp1, *d_temp2;

  char test;

  // Obtenim la dimensió de les matrius i comprovació de resultat
  if (argc == 3) { 
     N = atoi(argv[1]); 
     test = *argv[2];
  }
  else { printf("Usage: ./exe N test\n"); exit(0); }

  // Comprovem que N és potència de 2 (requeriment per a Strassen)
  if((N & (N-1)) != 0) {
    printf("N ha de ser potència de 2 per a l'algoritme de Strassen\n");
    exit(0);
  }

  // Nombre de fils en cada dimensió 
  nThreads = SIZE;

  // Nombre de blocs en cada dimensió 
  nBlocks = (N+nThreads-1)/nThreads;
  
  numBytes = N * N * sizeof(float);
  numBytesSubmatrix = (N/2) * (N/2) * sizeof(float);

  dim3 dimGrid(nBlocks, nBlocks, 1);
  dim3 dimGridHalf((N/2+nThreads-1)/nThreads, (N/2+nThreads-1)/nThreads, 1);
  dim3 dimBlock(nThreads, nThreads, 1);

  hipEventCreate(&E0);
  hipEventCreate(&E1);
  hipEventCreate(&E2);
  hipEventCreate(&E3);

  if (PINNED) {
    // Obtenim memòria pinned en el host
    hipHostMalloc((float**)&h_A, numBytes); 
    hipHostMalloc((float**)&h_B, numBytes); 
    hipHostMalloc((float**)&h_C, numBytes); 
  }
  else {
    // Obtenim memòria en el host
    h_A = (float*) malloc(numBytes); 
    h_B = (float*) malloc(numBytes); 
    h_C = (float*) malloc(numBytes); 
  }

  // Inicialitzem les matrius
  InitM(N, N, h_A);
  InitM(N, N, h_B);

  hipEventRecord(E0, 0);
  hipEventSynchronize(E0);
  
  // Obtenim memòria en el device per a les matrius completes
  hipMalloc((float**)&d_A, numBytes); 
  hipMalloc((float**)&d_B, numBytes); 
  hipMalloc((float**)&d_C, numBytes); 
  
  // Obtenim memòria per a les submatrius
  hipMalloc((float**)&d_A11, numBytesSubmatrix);
  hipMalloc((float**)&d_A12, numBytesSubmatrix);
  hipMalloc((float**)&d_A21, numBytesSubmatrix);
  hipMalloc((float**)&d_A22, numBytesSubmatrix);
  
  hipMalloc((float**)&d_B11, numBytesSubmatrix);
  hipMalloc((float**)&d_B12, numBytesSubmatrix);
  hipMalloc((float**)&d_B21, numBytesSubmatrix);
  hipMalloc((float**)&d_B22, numBytesSubmatrix);
  
  hipMalloc((float**)&d_C11, numBytesSubmatrix);
  hipMalloc((float**)&d_C12, numBytesSubmatrix);
  hipMalloc((float**)&d_C21, numBytesSubmatrix);
  hipMalloc((float**)&d_C22, numBytesSubmatrix);
  
  // Memòria per a les matrius temporals M
  hipMalloc((float**)&d_M1, numBytesSubmatrix);
  hipMalloc((float**)&d_M2, numBytesSubmatrix);
  hipMalloc((float**)&d_M3, numBytesSubmatrix);
  hipMalloc((float**)&d_M4, numBytesSubmatrix);
  hipMalloc((float**)&d_M5, numBytesSubmatrix);
  hipMalloc((float**)&d_M6, numBytesSubmatrix);
  hipMalloc((float**)&d_M7, numBytesSubmatrix);
  
  // Memòria per a matrius temporals auxiliars
  hipMalloc((float**)&d_temp1, numBytesSubmatrix);
  hipMalloc((float**)&d_temp2, numBytesSubmatrix);

  // Copiem dades des del host al device 
  hipMemcpy(d_A, h_A, numBytes, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, numBytes, hipMemcpyHostToDevice);

  // Kernel per dividir les matrius A i B en submatrius
  // Això no és un kernel, és un codi del host per dividir matrius i copiar-les al device
  float *h_A11, *h_A12, *h_A21, *h_A22;
  float *h_B11, *h_B12, *h_B21, *h_B22;
  
  h_A11 = (float*) malloc(numBytesSubmatrix);
  h_A12 = (float*) malloc(numBytesSubmatrix);
  h_A21 = (float*) malloc(numBytesSubmatrix);
  h_A22 = (float*) malloc(numBytesSubmatrix);
  
  h_B11 = (float*) malloc(numBytesSubmatrix);
  h_B12 = (float*) malloc(numBytesSubmatrix);
  h_B21 = (float*) malloc(numBytesSubmatrix);
  h_B22 = (float*) malloc(numBytesSubmatrix);
  
  // Dividim les matrius
  partitionMatrix(h_A, h_A11, h_A12, h_A21, h_A22, N);
  partitionMatrix(h_B, h_B11, h_B12, h_B21, h_B22, N);
  
  // Copiem les submatrius al device
  hipMemcpy(d_A11, h_A11, numBytesSubmatrix, hipMemcpyHostToDevice);
  hipMemcpy(d_A12, h_A12, numBytesSubmatrix, hipMemcpyHostToDevice);
  hipMemcpy(d_A21, h_A21, numBytesSubmatrix, hipMemcpyHostToDevice);
  hipMemcpy(d_A22, h_A22, numBytesSubmatrix, hipMemcpyHostToDevice);
  
  hipMemcpy(d_B11, h_B11, numBytesSubmatrix, hipMemcpyHostToDevice);
  hipMemcpy(d_B12, h_B12, numBytesSubmatrix, hipMemcpyHostToDevice);
  hipMemcpy(d_B21, h_B21, numBytesSubmatrix, hipMemcpyHostToDevice);
  hipMemcpy(d_B22, h_B22, numBytesSubmatrix, hipMemcpyHostToDevice);

  hipEventRecord(E1, 0);
  hipEventSynchronize(E1);
  
  // Algorisme de Strassen
  int halfN = N/2;
  
  // M1 = (A11 + A22) * (B11 + B22)
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_A11, d_A22, d_temp1);
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_B11, d_B22, d_temp2);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_temp1, d_temp2, d_M1);
  
  // M2 = (A21 + A22) * B11
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_A21, d_A22, d_temp1);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_temp1, d_B11, d_M2);
  
  // M3 = A11 * (B12 - B22)
  KernelSub<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_B12, d_B22, d_temp1);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_A11, d_temp1, d_M3);
  
  // M4 = A22 * (B21 - B11)
  KernelSub<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_B21, d_B11, d_temp1);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_A22, d_temp1, d_M4);
  
  // M5 = (A11 + A12) * B22
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_A11, d_A12, d_temp1);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_temp1, d_B22, d_M5);
  
  // M6 = (A21 - A11) * (B11 + B12)
  KernelSub<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_A21, d_A11, d_temp1);
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_B11, d_B12, d_temp2);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_temp1, d_temp2, d_M6);
  
  // M7 = (A12 - A22) * (B21 + B22)
  KernelSub<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_A12, d_A22, d_temp1);
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_B21, d_B22, d_temp2);
  KernelMult<<<dimGridHalf, dimBlock>>>(halfN, halfN, halfN, d_temp1, d_temp2, d_M7);
  
  // C11 = M1 + M4 - M5 + M7
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_M1, d_M4, d_temp1);
  KernelSub<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_temp1, d_M5, d_temp2);
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_temp2, d_M7, d_C11);
  
  // C12 = M3 + M5
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_M3, d_M5, d_C12);
  
  // C21 = M2 + M4
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_M2, d_M4, d_C21);
  
  // C22 = M1 - M2 + M3 + M6
  KernelSub<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_M1, d_M2, d_temp1);
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_temp1, d_M3, d_temp2);
  KernelAdd<<<dimGridHalf, dimBlock>>>(halfN, halfN, d_temp2, d_M6, d_C22);
  
  // Copiem les submatrius de la GPU al host
  float *h_C11, *h_C12, *h_C21, *h_C22;
  
  h_C11 = (float*) malloc(numBytesSubmatrix);
  h_C12 = (float*) malloc(numBytesSubmatrix);
  h_C21 = (float*) malloc(numBytesSubmatrix);
  h_C22 = (float*) malloc(numBytesSubmatrix);
  
  hipMemcpy(h_C11, d_C11, numBytesSubmatrix, hipMemcpyDeviceToHost);
  hipMemcpy(h_C12, d_C12, numBytesSubmatrix, hipMemcpyDeviceToHost);
  hipMemcpy(h_C21, d_C21, numBytesSubmatrix, hipMemcpyDeviceToHost);
  hipMemcpy(h_C22, d_C22, numBytesSubmatrix, hipMemcpyDeviceToHost);
  
  // Reconstruïm la matriu C a partir de les submatrius
  joinMatrix(h_C, h_C11, h_C12, h_C21, h_C22, N);
  
  // També copiem el resultat al device per si cal fer alguna altra operació
  hipMemcpy(d_C, h_C, numBytes, hipMemcpyHostToDevice);

  hipEventRecord(E2, 0);
  hipEventSynchronize(E2);
  
  // Alliberem la memòria del device 
  hipFree(d_A11); hipFree(d_A12); hipFree(d_A21); hipFree(d_A22);
  hipFree(d_B11); hipFree(d_B12); hipFree(d_B21); hipFree(d_B22);
  hipFree(d_C11); hipFree(d_C12); hipFree(d_C21); hipFree(d_C22);
  hipFree(d_M1); hipFree(d_M2); hipFree(d_M3); hipFree(d_M4); 
  hipFree(d_M5); hipFree(d_M6); hipFree(d_M7);
  hipFree(d_temp1); hipFree(d_temp2);

  hipEventRecord(E3, 0);
  hipEventSynchronize(E3);

  hipEventElapsedTime(&TiempoTotal, E0, E3);
  hipEventElapsedTime(&TiempoKernel, E1, E2);
  printf("\nALGORITME DE STRASSEN\n");
  printf("Dimensions: %dx%d\n", N, N);
  printf("nThreads: %dx%d (%d)\n", nThreads, nThreads, nThreads * nThreads);
  printf("nBlocks: %dx%d (%d)\n", nBlocks, nBlocks, nBlocks*nBlocks);
  if (PINNED) printf("Utilitzant Pinned Memory\n");
         else printf("NO utilitza Pinned Memory\n");
  printf("Temps Global: %4.6f milseg\n", TiempoTotal);
  printf("Temps Kernel: %4.6f milseg\n", TiempoKernel);
  printf("Rendiment Global: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoTotal));
  printf("Rendiment Kernel: %4.2f GFLOPS\n", (2.0 * (float) N * (float) N * (float) N) / (1000000.0 * TiempoKernel));

  hipEventDestroy(E0); hipEventDestroy(E1); hipEventDestroy(E2); hipEventDestroy(E3);

  if (test == 'N')
    printf ("NO TEST\n");
  else if (TestMM(N, N, N, h_A, h_B, h_C))
    printf ("TEST PASS\n");
  else
    printf ("TEST FAIL\n");

  // Alliberem la memòria del host
  free(h_A11); free(h_A12); free(h_A21); free(h_A22);
  free(h_B11); free(h_B12); free(h_B21); free(h_B22);
  free(h_C11); free(h_C12); free(h_C21); free(h_C22);
  
  if (PINNED) {
    hipHostFree(h_A); hipHostFree(h_B); hipHostFree(h_C);
  }
  else {
    free(h_A); free(h_B); free(h_C);
  }

  return 0;
}
